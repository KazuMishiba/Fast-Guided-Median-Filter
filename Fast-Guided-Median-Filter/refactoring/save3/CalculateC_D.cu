#include "hip/hip_runtime.h"
#include "CalculateC_D.cuh"

namespace FGMF_GPU_Or
{

//���E�g����
__global__ void
de_gsum_x(int width, int height, int radius, int2* sumG, hipTextureObject_t texG, size_t pitchI2)
{
	int y = blockIdx.x * blockDim.x + threadIdx.x;
	if (y >= height)
		return;

	int g;
	int _g;
	int sumg = 0;
	int sumgg = 0;

	//x=0
	for (int x = -radius; x <= radius; x++)
	{
		g = tex2D<int>(texG, x, y);
		sumg += g;
		sumgg += g * g;
	}
	*((int2*)((char*)sumG + y * pitchI2)) = make_int2(sumg, sumgg);

	for (int x = 1; x < width; x++)
	{
		g = tex2D<int>(texG, x + radius, y);
		_g = tex2D<int>(texG, x - radius - 1, y);
		sumg += g - _g;
		sumgg += g * g - _g * _g;
		*((int2*)((char*)sumG + y * pitchI2) + x) = make_int2(sumg, sumgg);
	}
}
__global__ void
de_gsum_y(int width, int height, int radius, int2* sumG, hipTextureObject_t texSumG, size_t pitchI2)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= width)
		return;

	int2 tmp, _tmp;
	int sumg = 0;
	int sumgg = 0;

	//y = 0
	for (int y = -radius; y <= radius; y++)
	{
		tmp = tex2D<int2>(texSumG, x, y);
		sumg += tmp.x;
		sumgg += tmp.y;
	}
	*((int2*)((char*)sumG) + x) = make_int2(sumg, sumgg);

	for (int y = 1; y < height; y++)
	{
		tmp = tex2D<int2>(texSumG, x, y + radius);
		_tmp = tex2D<int2>(texSumG, x, y - radius - 1);
		sumg += tmp.x - _tmp.x;
		sumgg += tmp.y - _tmp.y;
		*((int2*)((char*)sumG + y * pitchI2) + x) = make_int2(sumg, sumgg);
	}
}

//g3
__global__ void
de_g3sum_x(int width, int height, int radius, int** sumG, int** sumGG, hipTextureObject_t* texG3, size_t pitchI1)
{
	int y = blockIdx.x * blockDim.x + threadIdx.x;
	if (y >= height)
		return;

	int g[3];
	int _g[3];
	int sumg[3] = { 0, 0, 0 };
	int sumgg[6] = { 0,0,0,0,0,0 };//11,12,13,22,23,33

	//x=0
	for (int x = -radius; x <= radius; x++)
	{
		for (int i = 0; i < 3; i++)
			g[i] = tex2D<int>(texG3[i], x, y);

		for (int i = 0; i < 3; i++)
			sumg[i] += g[i];

		int n = 0;
		for (int i = 0; i < 3; i++) {
			for (int j = i; j < 3; j++) {
				sumgg[n] += g[i] * g[j];
				n++;
			}
		}
	}
	for (int i = 0; i < 3; i++)
		*((int*)((char*)(sumG[i]) + y * pitchI1)) = sumg[i];
	for (int i = 0; i < 6; i++)
		*((int*)((char*)sumGG[i] + y * pitchI1)) = sumgg[i];

	for (int x = 1; x < width; x++)
	{
		for (int i = 0; i < 3; i++)
			g[i] = tex2D<int>(texG3[i], x + radius, y);
		for (int i = 0; i < 3; i++)
			_g[i] = tex2D<int>(texG3[i], x - radius - 1, y);
		for (int i = 0; i < 3; i++)
			sumg[i] += g[i] - _g[i];
		int n = 0;
		for (int i = 0; i < 3; i++) {
			for (int j = i; j < 3; j++) {
				sumgg[n] += g[i] * g[j] - _g[i] * _g[j];
				n++;
			}
		}
		for (int i = 0; i < 3; i++)
			*((int*)((char*)sumG[i] + y * pitchI1) + x) = sumg[i];
		for (int i = 0; i < 6; i++)
			*((int*)((char*)sumGG[i] + y * pitchI1) + x) = sumgg[i];
	}
	
}


__global__ void
de_g3sum_y(int width, int height, int radius, int** sumG, int** sumGG, hipTextureObject_t* texSumG, hipTextureObject_t* texSumGG, size_t pitchI1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= width)
		return;
	
	int sumg[3] = { 0, 0, 0 };
	int sumgg[6] = { 0, 0, 0, 0, 0, 0 };

	//y = 0
	for (int y = -radius; y <= radius; y++)
	{
		for (int i = 0; i < 3; i++)
			sumg[i] += tex2D<int>(texSumG[i], x, y);
		for (int i = 0; i < 6; i++)
			sumgg[i] += tex2D<int>(texSumGG[i], x, y);

	}
	for (int i = 0; i < 3; i++)
		*((int*)((char*)sumG[i]) + x) = sumg[i];
	for (int i = 0; i < 6; i++)
		*((int*)((char*)sumGG[i]) + x) = sumgg[i];

	for (int y = 1; y < height; y++)
	{
		for (int i = 0; i < 3; i++)
			sumg[i] += tex2D<int>(texSumG[i], x, y + radius) - tex2D<int>(texSumG[i], x, y - radius - 1);
		for (int i = 0; i < 6; i++)
			sumgg[i] += tex2D<int>(texSumGG[i], x, y + radius) - tex2D<int>(texSumGG[i], x, y - radius - 1);

		for (int i = 0; i < 3; i++)
			*((int*)((char*)sumG[i] + y * pitchI1) + x) = sumg[i];
		for (int i = 0; i < 6; i++)
			*((int*)((char*)sumGG[i] + y * pitchI1) + x) = sumgg[i];
	}	
}


//gX
__global__ void
de_gXsum_x(int width, int height, int radius, int** sumG, int** sumGG, hipTextureObject_t* texGX, size_t pitchI1, int n)
{
	int y = blockIdx.x * blockDim.x + threadIdx.x;
	if (y >= height)
		return;

	int m = (n + 1)*n / 2;
	int *g = new int[n];
	int *_g = new int[n];
	int *sumg = new int[n];
	for (int i = 0; i < n; i++)
		sumg[i] = 0;
	int *sumgg = new int[m];
	for (int i = 0; i < m; i++)
		sumgg[i] = 0;


	//x=0
	for (int x = -radius; x <= radius; x++)
	{
		for (int i = 0; i < n; i++)
			g[i] = tex2D<int>(texGX[i], x, y);

		for (int i = 0; i < n; i++)
			sumg[i] += g[i];

		int k = 0;
		for (int i = 0; i < n; i++) {
			for (int j = i; j < n; j++) {
				sumgg[k] += g[i] * g[j];
				k++;
			}
		}
	}
	for (int i = 0; i < n; i++)
		*((int*)((char*)(sumG[i]) + y * pitchI1)) = sumg[i];
	for (int i = 0; i < m; i++)
		*((int*)((char*)sumGG[i] + y * pitchI1)) = sumgg[i];

	for (int x = 1; x < width; x++)
	{
		for (int i = 0; i < n; i++)
			g[i] = tex2D<int>(texGX[i], x + radius, y);
		for (int i = 0; i < n; i++)
			_g[i] = tex2D<int>(texGX[i], x - radius - 1, y);
		for (int i = 0; i < n; i++)
			sumg[i] += g[i] - _g[i];
		int k = 0;
		for (int i = 0; i < n; i++) {
			for (int j = i; j < n; j++) {
				sumgg[k] += g[i] * g[j] - _g[i] * _g[j];
				k++;
			}
		}
		for (int i = 0; i < n; i++)
			*((int*)((char*)sumG[i] + y * pitchI1) + x) = sumg[i];
		for (int i = 0; i < m; i++)
			*((int*)((char*)sumGG[i] + y * pitchI1) + x) = sumgg[i];
	}

	delete g;
	delete _g;
	delete sumg;
	delete sumgg;

}

__global__ void
de_gXsum_y(int width, int height, int radius, int** sumG, int** sumGG, hipTextureObject_t* texSumG, hipTextureObject_t* texSumGG, size_t pitchI1, int n)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= width)
		return;

	int m = (n + 1)*n / 2;
	int *sumg = new int[n];
	for (int i = 0; i < n; i++)
		sumg[i] = 0;
	int *sumgg = new int[m];
	for (int i = 0; i < m; i++)
		sumgg[i] = 0;


	//y = 0
	for (int y = -radius; y <= radius; y++)
	{
		for (int i = 0; i < n; i++)
			sumg[i] += tex2D<int>(texSumG[i], x, y);
		for (int i = 0; i < m; i++)
			sumgg[i] += tex2D<int>(texSumGG[i], x, y);

	}
	for (int i = 0; i < n; i++)
		*((int*)((char*)sumG[i]) + x) = sumg[i];
	for (int i = 0; i < m; i++)
		*((int*)((char*)sumGG[i]) + x) = sumgg[i];

	for (int y = 1; y < height; y++)
	{
		for (int i = 0; i < n; i++)
			sumg[i] += tex2D<int>(texSumG[i], x, y + radius) - tex2D<int>(texSumG[i], x, y - radius - 1);
		for (int i = 0; i < m; i++)
			sumgg[i] += tex2D<int>(texSumGG[i], x, y + radius) - tex2D<int>(texSumGG[i], x, y - radius - 1);

		for (int i = 0; i < n; i++)
			*((int*)((char*)sumG[i] + y * pitchI1) + x) = sumg[i];
		for (int i = 0; i < m; i++)
			*((int*)((char*)sumGG[i] + y * pitchI1) + x) = sumgg[i];
	}

	delete sumg;
	delete sumgg;
}



//cx,dx float2
__global__ void
de_calculateDC(int width, int height, int* G, int2* sumG, float eps2, float pixel_sum_window_inv, float2* dxcx, size_t pitchI1, size_t pitchI2, size_t pitchF2)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width || y < 0 || y >= height)
		return;

	int2 tmp = *((int2*)((char*)sumG + y * pitchI2) + x);
	int g = *((int*)((char*)G + y * pitchI1) + x);
	float g_ave = ((float)tmp.x) * pixel_sum_window_inv;
	float gg_ave = ((float)tmp.y) * pixel_sum_window_inv;
	float vx = gg_ave - g_ave * g_ave + eps2;
	float tmp2 = ((float)g) - g_ave;
	float cx2 = tmp2 * pixel_sum_window_inv / vx;
	*((float2*)((char*)dxcx + y * pitchF2) + x) = make_float2(pixel_sum_window_inv - g_ave * cx2, cx2);
}
//cx3
__global__ void
de_calculateCx3Dx(int width, int height, int** G3, int** sumG, int** sumGG, float eps2, float pixel_sum_window_inv, float4* dxcx, size_t pitchI1, size_t pitchF4)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width || y < 0 || y >= height)
		return;

	float g_ave1 = *((int*)((char*)sumG[0] + y * pitchI1) + x) * pixel_sum_window_inv;
	float g_ave2 = *((int*)((char*)sumG[1] + y * pitchI1) + x) * pixel_sum_window_inv;
	float g_ave3 = *((int*)((char*)sumG[2] + y * pitchI1) + x) * pixel_sum_window_inv;
	float v11 = *((int*)((char*)sumGG[0] + y * pitchI1) + x) * pixel_sum_window_inv - g_ave1 * g_ave1 + eps2;
	float v12 = *((int*)((char*)sumGG[1] + y * pitchI1) + x) * pixel_sum_window_inv - g_ave1 * g_ave2;
	float v13 = *((int*)((char*)sumGG[2] + y * pitchI1) + x) * pixel_sum_window_inv - g_ave1 * g_ave3;
	float v22 = *((int*)((char*)sumGG[3] + y * pitchI1) + x) * pixel_sum_window_inv - g_ave2 * g_ave2 + eps2;
	float v23 = *((int*)((char*)sumGG[4] + y * pitchI1) + x) * pixel_sum_window_inv - g_ave2 * g_ave3;
	float v33 = *((int*)((char*)sumGG[5] + y * pitchI1) + x) * pixel_sum_window_inv - g_ave3 * g_ave3 + eps2;
	float delta =
		v11 * v22 * v33 +
		v12 * v23 * v13 * 2 -
		v13 * v13 * v22 -
		v12 * v12 * v33 -
		v11 * v23 * v23;
	if (abs(delta) > 0.000001f)
	{
		float deltaInv = 1.0f / delta;
		float vinv11 = (v22 * v33 - v23 * v23);
		float vinv12 = (v13 * v23 - v12 * v33);
		float vinv13 = (v12 * v23 - v13 * v22);
		float vinv22 = (v11 * v33 - v13 * v13);
		float vinv23 = (v13 * v12 - v11 * v23);
		float vinv33 = (v11 * v22 - v12 * v12);
		float tmp1 = *((int*)((char*)G3[0] + y * pitchI1) + x) - g_ave1;
		float tmp2 = *((int*)((char*)G3[1] + y * pitchI1) + x) - g_ave2;
		float tmp3 = *((int*)((char*)G3[2] + y * pitchI1) + x) - g_ave3;
		float mult = pixel_sum_window_inv * deltaInv;
		float cx1 = (tmp1 * vinv11 + tmp2 * vinv12 + tmp3 * vinv13) * mult;
		float cx2 = (tmp1 * vinv12 + tmp2 * vinv22 + tmp3 * vinv23) * mult;
		float cx3 = (tmp1 * vinv13 + tmp2 * vinv23 + tmp3 * vinv33) * mult;
		float dx = pixel_sum_window_inv - g_ave1 * cx1 - g_ave2 * cx2 - g_ave3 * cx3;

		*((float4*)((char*)dxcx + y * pitchF4) + x) = make_float4(dx, cx1, cx2, cx3);
	}
	else
	{
		//�t�s�񂪑��݂��Ȃ��̂ŁAcx=0, dx=��f���̋t���Ƃ���
		*((float4*)((char*)dxcx + y * pitchF4) + x) = make_float4(pixel_sum_window_inv, 0.0f, 0.0f, 0.0f);
	}
}
//cxX
__global__ void
de_calculateCxXDx(int width, int height, int** GX, int** sumG, int** sumGG, float eps2, float pixel_sum_window_inv, float** dxcx, size_t pitchI1, size_t pitchF1, int n)
{
	//dxcx�̗v�f����N+1 (cx��N�Adx��1)
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width || y < 0 || y >= height)
		return;
	


	int m = (n + 1)*n / 2;

	float *g_ave = new float[n];
	for (int i = 0; i < n; i++)
		g_ave[i] = *((int*)((char*)sumG[i] + y * pitchI1) + x) * pixel_sum_window_inv;
	float *A = new float[m];
	int k = 0;
	for (int j = 0; j < n; j++) {
		A[k] = *((int*)((char*)sumGG[k] + y * pitchI1) + x) * pixel_sum_window_inv - g_ave[j] * g_ave[j] + eps2;
		k++;
		for (int i = j + 1; i < n; i++) {
			A[k] = *((int*)((char*)sumGG[k] + y * pitchI1) + x) * pixel_sum_window_inv - g_ave[i] * g_ave[j];
			k++;
		}
	}



	//�������z�@
	//�����l0
	//�������m��
	float *cx = new float[n];
	float *r = new float[n];
	float *p = new float[n];
	float *Ap = new float[n];
	float rsold = 0.0f;
	float alpha, rsnew;

	bool flag = true;

	for (int i = 0; i < n; i++)
	{
		r[i] = *((int*)((char*)GX[i] + y * pitchI1) + x) - g_ave[i];
		p[i] = r[i];
		rsold += r[i] * r[i];
		cx[i] = 0.0f;

		flag = flag & (r[i] == 0.0f);
	}



	if (!flag)
	{
		for (int iter = 0; iter < n; iter++)
		{
			alpha = 0.0f;
			//Ap = A * p
			int t = 0;
			for (int j = 0; j < n; j++)
			{
				//�Ίp�̉��Z���ʂŏ�����
				//i == j
				Ap[j] = A[t] * p[j];

				//�~���[����
				int m = j;
				int d = n - 1;
				for (int i = 0; i < j; i++)
				{
					Ap[j] += A[m] * p[i];
					m += d;
					d--;
				}
				//
				for (int i = j + 1; i < n; i++)
				{
					t++;
					Ap[j] += A[t] * p[i];
				}
				t++;

				//alpha = rsold / (p' * Ap);
				alpha += p[j] * Ap[j];
			}

			alpha = rsold / alpha;

			rsnew = 0.0f;
			for (int i = 0; i < n; i++)
			{
				//x = x + alpha * p;
				cx[i] += alpha * p[i];
				//r = r - alpha * Ap;
				r[i] -= alpha * Ap[i];
				//rsnew = r' * r;
				rsnew += r[i] * r[i];
			}
			if (rsnew < 0.000000000000001)
			{
				//printf("b:");
				break;
			}
			//p = r + (rsnew / rsold) * p;
			float no = rsnew / rsold;
			for (int i = 0; i < n; i++)
			{
				p[i] = r[i] + no * p[i];
			}
			//rsold = rsnew;
			rsold = rsnew;

		}

		float dx = pixel_sum_window_inv;
		*((float*)((char*)dxcx[0] + y * pitchF1) + x) = dx;
		for (int i = 0; i < n; i++)
		{
			cx[i] *= pixel_sum_window_inv;
			dx -= cx[i] * g_ave[i];
			*((float*)((char*)dxcx[i+1] + y * pitchF1) + x) = cx[i];
		}
	}
	else
	{
		//�t�s�񂪑��݂��Ȃ��ꍇ
		//cx�S�ĂO�Adx�͉�f���̋t���i���ʂƂ��Ă͕��ϒl�t�B���^�J�[�l���Ɠ����j
		*((float*)((char*)dxcx[0] + y * pitchF1) + x) = pixel_sum_window_inv;
		for (int i = 0; i < n; i++)
		{
			*((float*)((char*)dxcx[i + 1] + y * pitchF1) + x) = 0.0f;
		}
	}

	//if (isnan(cx[0]))
	{
		/*
		printf("\n");

		printf("g:\n");
		for (int i = 0; i < n; i++)
			printf("%f ", *((int*)((char*)GX[i] + y * pitchI1) + x));
		printf("\n");
		*/
		/*
		printf("gave:\n");
		for (int i = 0; i < n; i++)
			printf("%f ", g_ave[i]);
		printf("\n");

		printf("g-gave:\n");
		for (int i = 0; i < n; i++)
			printf("%f ", *((int*)((char*)GX[i] + y * pitchI1) + x) - g_ave[i]);
		printf("\n");

		printf("cx:\n");
		for (int i = 0; i < n; i++)
			printf("%f ", cx[i]);
		printf("\n");
		*/
	}

	delete g_ave;
	delete cx;
	delete A;
	delete r;
	delete p;
	delete Ap;






}

/*
�������z�@��������
MATLAB�R�[�h
function x = conjgrad(A, b, x)
	r = b - A * x;
	p = r;
	rsold = r' * r;

	for i = 1:length(b)
		Ap = A * p;
		alpha = rsold / (p' * Ap);
		x = x + alpha * p;
		r = r - alpha * Ap;
		rsnew = r' * r;
		if sqrt(rsnew) < 1e-10
			  break
		end
		p = r + (rsnew / rsold) * p;
		rsold = rsnew;
	end
end

%A = a * a' + c I�̂Ƃ��̓���
function x = conjgrad2(a, b, c, x)
	r = b - a * (a' * x) - c * x;
	p = r;
	rsold = r' * r;

	for i = 1:length(b)
		Ap = a * (a' * p) + c * p;
		alpha = rsold / (p' * Ap);
		x = x + alpha * p;
		r = r - alpha * Ap;
		rsnew = r' * r;
		if sqrt(rsnew) < 1e-10
			  break
		end
		p = r + (rsnew / rsold) * p;
		rsold = rsnew;
	end
end
����̓E�B���h�E���a���̉�f���L�^���Ȃ��Ƃ����Ȃ��̂ō���g���Ȃ��B

*/


//sumG(sumg, sumgg, pixel_num, g) ���v�Z
void cu_calculateSumG(Helper::SizeInfo& sizeInfo, hipStream_t stream, int* G, int radius, int2* sumG, int2* temp)
{
	int blockSize = BLOCK_SIZE_1D;
	int gridSizeY = ceil(sizeInfo.height_ / (float)blockSize);
	int gridSizeX = ceil(sizeInfo.width_ / (float)blockSize);

	hipTextureObject_t texG;
	hipTextureFilterMode filterMode = hipTextureFilterMode::hipFilterModePoint;
	//texG��G���o�C���h
	Helper::UtilityForCUDA::setLinearArrayToTexture(G, texG, sizeInfo, filterMode);
	//texG�̓��e����sumG�Ȃǌv�Z��temp�Ɋi�[
	de_gsum_x << < gridSizeY, blockSize, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, radius, temp, texG, sizeInfo.pitch<int2>());
	//temp��texG�Ƀo�C���h���A�c������sumG�Ȃǌv�Z��sumG�Ɋi�[
	Helper::UtilityForCUDA::setLinearArrayToTexture(temp, texG, sizeInfo, filterMode);
	//texG�̓��e����sumG�Ȃǌv�Z��sumG�Ɋi�[
	de_gsum_y << < gridSizeX, blockSize, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, radius, sumG, texG, sizeInfo.pitch<int2>());

	hipDestroyTextureObject(texG);
}
//g3
void cu_calculateSumG3(Helper::SizeInfo& sizeInfo, hipStream_t stream, Helper::DeviceArray<int>* G3, int radius, Helper::DeviceArray<int>* sumG, Helper::DeviceArray<int>* sumGG, Helper::DeviceArray<int>* tempG, Helper::DeviceArray<int>* tempGG)
{
	int blockSize = BLOCK_SIZE_1D;
	int gridSizeY = ceil(sizeInfo.height_ / (float)blockSize);
	int gridSizeX = ceil(sizeInfo.width_ / (float)blockSize);

	hipTextureFilterMode filterMode = hipTextureFilterMode::hipFilterModePoint;
	Helper::TextureArray<int>* texG = new Helper::TextureArray<int>(G3, filterMode, sizeInfo);
	//texG�̓��e����gsum��tempG�ɁAggsum��tempGG�Ɋi�[
	de_g3sum_x << < gridSizeY, blockSize, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, radius, tempG->device, tempGG->device, texG->device, sizeInfo.pitch<int>());
	//tempG��texSumG�ɁAtempGG��texSumGG�Ƀo�C���h
	Helper::TextureArray<int>* texSumG = new Helper::TextureArray<int>(tempG, filterMode, sizeInfo);
	Helper::TextureArray<int>* texSumGG = new Helper::TextureArray<int>(tempGG, filterMode, sizeInfo);
	//sumG�AsumGG�v�Z
	de_g3sum_y << < gridSizeX, blockSize, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, radius, sumG->device, sumGG->device, texSumG->device, texSumGG->device, sizeInfo.pitch<int>());
	
	delete texG;
	delete texSumG;
	delete texSumGG;
}
//gX
void cu_calculateSumGX(Helper::SizeInfo& sizeInfo, hipStream_t stream, Helper::DeviceArray<int>* GX, int radius, Helper::DeviceArray<int>* sumG, Helper::DeviceArray<int>* sumGG, Helper::DeviceArray<int>* tempG, Helper::DeviceArray<int>* tempGG, int n)
{
	int blockSize = BLOCK_SIZE_1D;
	int gridSizeY = ceil(sizeInfo.height_ / (float)blockSize);
	int gridSizeX = ceil(sizeInfo.width_ / (float)blockSize);

	hipTextureFilterMode filterMode = hipTextureFilterMode::hipFilterModePoint;
	Helper::TextureArray<int>* texG = new Helper::TextureArray<int>(GX, filterMode, sizeInfo);
	//texG�̓��e����gsum��tempG�ɁAggsum��tempGG�Ɋi�[
	de_gXsum_x << < gridSizeY, blockSize, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, radius, tempG->device, tempGG->device, texG->device, sizeInfo.pitch<int>(), n);
	//tempG��texSumG�ɁAtempGG��texSumGG�Ƀo�C���h
	Helper::TextureArray<int>* texSumG = new Helper::TextureArray<int>(tempG, filterMode, sizeInfo);
	Helper::TextureArray<int>* texSumGG = new Helper::TextureArray<int>(tempGG, filterMode, sizeInfo);
	//sumG�AsumGG�v�Z
	de_gXsum_y << < gridSizeX, blockSize, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, radius, sumG->device, sumGG->device, texSumG->device, texSumGG->device, sizeInfo.pitch<int>(), n);

	delete texG;
	delete texSumG;
	delete texSumGG;
}






//sumg���v�Z 2D�p cx,dx��float2�Œ�`
void cu_calculateDC(Helper::SizeInfo& sizeInfo, hipStream_t stream, int* G, int radius, int pixelNumInWindow, float eps2, float2* dxcx, int2* sumG, int2* temp)
{
	cu_calculateSumG(sizeInfo, stream, G, radius, sumG, temp);
	float pixel_sum_window_inv = 1.0f / pixelNumInWindow; ((radius * 2 + 1) * (radius * 2 + 1));
	//sumG�̒l����cx, dx���v�Z
	de_calculateDC << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, G, sumG, eps2, pixel_sum_window_inv, dxcx, sizeInfo.pitch<int>(), sizeInfo.pitch<int2>(), sizeInfo.pitch<float2>());
}
//g3
void cu_calculateDC3(Helper::SizeInfo& sizeInfo, hipStream_t stream, Helper::DeviceArray<int>* G3, int radius, int pixelNumInWindow, float eps2, float4* dxcx, Helper::DeviceArray<int>* sumG, Helper::DeviceArray<int>* sumGG, Helper::DeviceArray<int>* tempG, Helper::DeviceArray<int>* tempGG)
{
	cu_calculateSumG3(sizeInfo, stream, G3, radius, sumG, sumGG, tempG, tempGG);
	float pixel_sum_window_inv = 1.0f / pixelNumInWindow;
	//sumG�̒l����cx, dx���v�Z
	de_calculateCx3Dx << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, G3->device, sumG->device, sumGG->device, eps2, pixel_sum_window_inv, dxcx, sizeInfo.pitch<int>(), sizeInfo.pitch<float4>());
}

//gX
void cu_calculateCxXDxFromG(Helper::SizeInfo& sizeInfo, hipStream_t stream, Helper::DeviceArray<int>* GX, int radius, int pixelNumInWindow, float eps2, Helper::DeviceArray<float>* dxcx, Helper::DeviceArray<int>* sumG, Helper::DeviceArray<int>* sumGG, Helper::DeviceArray<int>* tempG, Helper::DeviceArray<int>* tempGG)
{
	int n = GX->arrayLength;
	cu_calculateSumGX(sizeInfo, stream, GX, radius, sumG, sumGG, tempG, tempGG, n);
	float pixel_sum_window_inv = 1.0f / pixelNumInWindow;
	//sumG�̒l����cx, dx���v�Z
	//���^�������������Ă��邱�̕������قƂ�ǂ̎��Ԃ�����Ă���
	de_calculateCxXDx << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, GX->device, sumG->device, sumGG->device, eps2, pixel_sum_window_inv, dxcx->device, sizeInfo.pitch<int>(), sizeInfo.pitch<float>(), n);

	//Utility::showDevice(dxcx->host[0], sizeInfo, "dxcx", true, 100000.0f);
}

//////////////////////////////////////////////////////////
// 3D�ȏ�p


//�ǉ��ƍ폜
__global__ void
de_updateSumG(int width, int height, int2* addSumG, int2* remSumG, int2* sumG, size_t pitchI2)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width || y < 0 || y >= height)
		return;

	int2 addsumg = *((int2*)((char*)addSumG + y * pitchI2) + x);
	int2 remsumg = *((int2*)((char*)remSumG + y * pitchI2) + x);
	int2 sumg = *((int2*)((char*)sumG + y * pitchI2) + x);
	*((int2*)((char*)sumG + y * pitchI2) + x) = make_int2(
		sumg.x + addsumg.x - remsumg.x,
		sumg.y + addsumg.y - remsumg.y
	);
}
__global__ void
de_updateSumG3(int width, int height, int** addSumG, int** addSumGG, int** remSumG, int** remSumGG, int** sumG, int** sumGG, size_t pitchI1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width || y < 0 || y >= height)
		return;

	for (int i = 0; i < 3; i++)
	{
		int addsumg = *((int*)((char*)addSumG[i] + y * pitchI1) + x);
		int remsumg = *((int*)((char*)remSumG[i] + y * pitchI1) + x);
		int sumg = *((int*)((char*)sumG[i] + y * pitchI1) + x);
		*((int*)((char*)sumG[i] + y * pitchI1) + x) = sumg + addsumg - remsumg;
	}
	for (int i = 0; i < 6; i++)
	{
		int addsumgg = *((int*)((char*)addSumGG[i] + y * pitchI1) + x);
		int remsumgg = *((int*)((char*)remSumGG[i] + y * pitchI1) + x);
		int sumgg = *((int*)((char*)sumGG[i] + y * pitchI1) + x);
		*((int*)((char*)sumGG[i] + y * pitchI1) + x) = sumgg + addsumgg - remsumgg;
	}
}
//�ǉ��̂�
__global__ void
de_addSumG(int width, int height, int2* addSumG, int2* sumG, size_t pitchI2)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width || y < 0 || y >= height)
		return;

	int2 addsumg = *((int2*)((char*)addSumG + y * pitchI2) + x);
	int2 sumg = *((int2*)((char*)sumG + y * pitchI2) + x);
	*((int2*)((char*)sumG + y * pitchI2) + x) = make_int2(
		sumg.x + addsumg.x,
		sumg.y + addsumg.y
	);
}
__global__ void
de_addSumG3(int width, int height, int** addSumG, int** addSumGG, int** sumG, int** sumGG, size_t pitchI1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width || y < 0 || y >= height)
		return;

	for (int i = 0; i < 3; i++)
	{
		int addsumg = *((int*)((char*)addSumG[i] + y * pitchI1) + x);
		int sumg = *((int*)((char*)sumG[i] + y * pitchI1) + x);
		*((int*)((char*)sumG[i] + y * pitchI1) + x) = sumg + addsumg;
	}
	for (int i = 0; i < 6; i++)
	{
		int addsumgg = *((int*)((char*)addSumGG[i] + y * pitchI1) + x);
		int sumgg = *((int*)((char*)sumGG[i] + y * pitchI1) + x);
		*((int*)((char*)sumGG[i] + y * pitchI1) + x) = sumgg + addsumgg;
	}
}
//�폜�̂�
__global__ void
de_remSumG(int width, int height, int2* remSumG, int2* sumG, size_t pitchI2)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width || y < 0 || y >= height)
		return;

	int2 remsumg = *((int2*)((char*)remSumG + y * pitchI2) + x);
	int2 sumg = *((int2*)((char*)sumG + y * pitchI2) + x);
	*((int2*)((char*)sumG + y * pitchI2) + x) = make_int2(
		sumg.x - remsumg.x,
		sumg.y - remsumg.y
	);
}
__global__ void
de_remSumG3(int width, int height, int** remSumG, int** remSumGG, int** sumG, int** sumGG, size_t pitchI1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width || y < 0 || y >= height)
		return;

	for (int i = 0; i < 3; i++)
	{
		int remsumg = *((int*)((char*)remSumG[i] + y * pitchI1) + x);
		int sumg = *((int*)((char*)sumG[i] + y * pitchI1) + x);
		*((int*)((char*)sumG[i] + y * pitchI1) + x) = sumg - remsumg;
	}
	for (int i = 0; i < 6; i++)
	{
		int remsumgg = *((int*)((char*)remSumGG[i] + y * pitchI1) + x);
		int sumgg = *((int*)((char*)sumGG[i] + y * pitchI1) + x);
		*((int*)((char*)sumGG[i] + y * pitchI1) + x) = sumgg - remsumgg;
	}
}

//update
void cu_updateSumG(Helper::SizeInfo& sizeInfo, hipStream_t stream, int* G, int radius, int2* addSumG, int2* remSumG, int2* sumG, int2* temp)
{
	cu_calculateSumG(sizeInfo, stream, G, radius, addSumG, temp);
	de_updateSumG << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, addSumG, remSumG, sumG, sizeInfo.pitch<int2>());
}
//g3
void cu_updateSumG3(Helper::SizeInfo& sizeInfo, hipStream_t stream, Helper::DeviceArray<int>* G, int radius, Helper::DeviceArray<int>* addSumG, Helper::DeviceArray<int>* addSumGG, Helper::DeviceArray<int>* remSumG, Helper::DeviceArray<int>* remSumGG, Helper::DeviceArray<int>* sumG, Helper::DeviceArray<int>* sumGG, Helper::DeviceArray<int>* tempG, Helper::DeviceArray<int>* tempGG)
{
	cu_calculateSumG3(sizeInfo, stream, G, radius, addSumG, addSumGG, tempG, tempGG);
	de_updateSumG3 << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, addSumG->device, addSumGG->device, remSumG->device, remSumGG->device, sumG->device, sumGG->device, sizeInfo.pitch<int>());
}

//add
void cu_addSumG(Helper::SizeInfo& sizeInfo, hipStream_t stream, int* G, int radius, int2* addSumG, int2* sumG, int2* temp)
{
	cu_calculateSumG(sizeInfo, stream, G, radius, addSumG, temp);
	de_addSumG << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, addSumG, sumG, sizeInfo.pitch<int2>());
}
//g3
void cu_addSumG3(Helper::SizeInfo& sizeInfo, hipStream_t stream, Helper::DeviceArray<int>* G, int radius, Helper::DeviceArray<int>* addSumG, Helper::DeviceArray<int>* addSumGG, Helper::DeviceArray<int>* sumG, Helper::DeviceArray<int>* sumGG, Helper::DeviceArray<int>* tempG, Helper::DeviceArray<int>* tempGG)
{
	cu_calculateSumG3(sizeInfo, stream, G, radius, addSumG, addSumGG, tempG, tempGG);
	de_addSumG3 << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, addSumG->device, addSumGG->device, sumG->device, sumGG->device, sizeInfo.pitch<int>());

}

//rem
void cu_remSumG(Helper::SizeInfo& sizeInfo, hipStream_t stream, int2* remSumG, int2* sumG)
{
	de_remSumG << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, remSumG, sumG, sizeInfo.pitch<int2>());
}
//g3
void cu_remSumG3(Helper::SizeInfo& sizeInfo, hipStream_t stream, Helper::DeviceArray<int>* remSumG, Helper::DeviceArray<int>* remSumGG, Helper::DeviceArray<int>* sumG, Helper::DeviceArray<int>* sumGG)
{
	de_remSumG3 << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, remSumG->device, remSumGG->device, sumG->device, sumGG->device, sizeInfo.pitch<int>());
}

//dxcx
void cu_calculateCxDx(Helper::SizeInfo& sizeInfo, hipStream_t stream, int* G, int radius, int pixelNumInWindow, float eps2, float2* dxcx, int2* sumG)
{
	float pixel_sum_window_inv = 1.0f / pixelNumInWindow;
	//sumG�̒l����cx, dx���v�Z
	de_calculateDC << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, G, sumG, eps2, pixel_sum_window_inv, dxcx, sizeInfo.pitch<int>(), sizeInfo.pitch<int2>(), sizeInfo.pitch<float2>());
}
//g3
void cu_calculateCx3Dx(Helper::SizeInfo& sizeInfo, hipStream_t stream, Helper::DeviceArray<int>* G, int radius, int pixelNumInWindow, float eps2, float4* dxcx, Helper::DeviceArray<int>* sumG, Helper::DeviceArray<int>* sumGG)
{
	float pixel_sum_window_inv = 1.0f / pixelNumInWindow;
	//sumG�̒l����cx, dx���v�Z
	de_calculateCx3Dx << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, G->device, sumG->device, sumGG->device, eps2, pixel_sum_window_inv, dxcx, sizeInfo.pitch<int>(), sizeInfo.pitch<float4>());
}



/*
//sumg���v�Z 3D�p �ǉ��ƍ폜���s���Ł@addSumG�ɒǉ�����sumg���v�Z���A����sumG�ɉ�����ƂƂ���remSumG���폜���� addSumG�͊m�ۂ��ꂽ��̂��̂�n���AremSumG�͒��g�̂���폜������n��
void cu_updateCxDx(Helper::SizeInfo& sizeInfo, hipStream_t stream, int* G, int radius, float eps2, float* cx, float* dx, int2* addSumG, int2* remSumG, int2* sumG, int2* temp)
{
	cu_calculateSumG(sizeInfo, stream, G, radius, addSumG, temp);
	//addSumG�̒ǉ���remSumG�̍폜
	de_updateSumG << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, addSumG, remSumG, sumG, sizeInfo.pitch<int>(), sizeInfo.pitch<int2>());
	//sumG�̒l����cx, dx���v�Z
	de_calculateCxDx << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, G, sumG, eps2, cx, dx, sizeInfo.pitch<int>(), sizeInfo.pitch<int2>(), sizeInfo.pitch<float>());
}
//�ǉ��̂�
void cu_updateCxDx_add(Helper::SizeInfo& sizeInfo, hipStream_t stream, int* G, int radius, float eps2, float* cx, float* dx, int2* addSumG, int2* sumG, int2* temp)
{
	cu_calculateSumG(sizeInfo, stream, G, radius, addSumG, temp);
	//addSumG�̒ǉ�
	de_addSumG << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, addSumG, sumG, sizeInfo.pitch<int>(), sizeInfo.pitch<int2>());
	//sumG�̒l����cx, dx���v�Z
	de_calculateCxDx << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, G, sumG, eps2, cx, dx, sizeInfo.pitch<int>(), sizeInfo.pitch<int2>(), sizeInfo.pitch<float>());
}
//�폜�̂�
void cu_updateCxDx_rem(Helper::SizeInfo& sizeInfo, hipStream_t stream, int* G, int radius, float eps2, float* cx, float* dx, int2* remSumG, int2* sumG, int2* temp)
{
	//remSumG�̍폜
	de_remSumG << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, remSumG, sumG, sizeInfo.pitch<int>(), sizeInfo.pitch<int2>());
	//sumG�̒l����cx, dx���v�Z
	de_calculateCxDx << <sizeInfo.gridSize_, sizeInfo.blockSize_, 0, stream >> > (sizeInfo.width_, sizeInfo.height_, G, sumG, eps2, cx, dx, sizeInfo.pitch<int>(), sizeInfo.pitch<int2>(), sizeInfo.pitch<float>());
}
*/

#if 0
//���E�g�����Ȃ���
//���E�g�����Ȃ���
__global__ void
de_gsum_x(int width_, int height_, int radius, int4* sumG, hipTextureObject_t texG, size_t pitchI4)
{
	int y = blockIdx.x * blockDim.x + threadIdx.x;
	if (y >= height_)
		return;


	int g;
	int _g;
	int sumg = 0.0f;
	int sumgg = 0.0f;
	int pixNum = radius + 1;

	//x=0
	for (int x = 0; x <= radius; x++)
	{
		g = tex2D<int>(texG, float(x) + 0.5f, float(y) + 0.5f);
		sumg += g;
		sumgg += g * g;
	}
	*((int4*)((char*)sumG + y * pitchI4)) = make_int4(sumg, sumgg, pixNum, g);

	int x = 1;
	//x=1~radius
	for (; x <= radius; x++)
	{
		g = tex2D<int>(texG, float(x + radius) + 0.5f, float(y) + 0.5f);
		sumg += g;
		sumgg += g * g;
		pixNum++;
		*((int4*)((char*)sumG + y * pitchI4) + x) = make_int4(sumg, sumgg, pixNum, g);
	}
	//x+radius��width-1�ɂȂ�܂ŉ����Z����
	//x = width - 1 - radius
	int bound = width_ - 1 - radius;
	for (; x <= bound; x++)
	{
		g = tex2D<int>(texG, float(x + radius) + 0.5f, float(y) + 0.5f);
		_g = tex2D<int>(texG, float(x - radius - 1) + 0.5f, float(y) + 0.5f);
		sumg += g - _g;
		sumgg += g * g - _g * _g;
		*((int4*)((char*)sumG + y * pitchI4) + x) = make_int4(sumg, sumgg, pixNum, g);
	}
	//x= ~width-1
	for (; x < width_; x++)
	{
		_g = tex2D<int>(texG, float(x - radius - 1) + 0.5f, float(y) + 0.5f);
		sumg -= _g;
		sumgg -= _g * _g;
		pixNum--;
		*((int4*)((char*)sumG + y * pitchI4) + x) = make_int4(sumg, sumgg, pixNum, g);
	}
}

__global__ void
de_gsum_y(int width_, int height_, int radius, int4* sumG, hipTextureObject_t texSumG, size_t pitchI4)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= width_)
		return;

	int4 tmp, _tmp;
	int sumg = 0.0f;
	int sumgg = 0.0f;
	int pixNum = 0;

	//y = 0
	for (int y = 0; y <= radius; y++)
	{
		tmp = tex2D<int4>(texSumG, float(x) + 0.5f, float(y) + 0.5f);
		sumg += tmp.x;
		sumgg += tmp.y;
		pixNum += tmp.z;
	}
	*((int4*)((char*)sumG) + x) = make_int4(sumg, sumgg, pixNum, sumg / pixNum);

	int y = 1;
	//y=1~radius
	for (; y <= radius; y++)
	{
		tmp = tex2D<int4>(texSumG, float(x) + 0.5f, float(y + radius) + 0.5f);
		sumg += tmp.x;
		sumgg += tmp.y;
		pixNum += tmp.z;
		*((int4*)((char*)sumG + y * pitchI4) + x) = make_int4(sumg, sumgg, pixNum, sumg / pixNum);
	}
	int bound = height_ - 1 - radius;
	for (; y < bound; y++)
	{
		tmp = tex2D<int4>(texSumG, float(x) + 0.5f, float(y + radius) + 0.5f);
		_tmp = tex2D<int4>(texSumG, float(x) + 0.5f, float(y - radius - 1) + 0.5f);
		sumg += tmp.x - _tmp.x;
		sumgg += tmp.y - _tmp.y;
		*((int4*)((char*)sumG + y * pitchI4) + x) = make_int4(sumg, sumgg, pixNum, sumg / pixNum);
	}
	for (; y < height_; y++)
	{
		_tmp = tex2D<int4>(texSumG, float(x) + 0.5f, float(y - radius - 1) + 0.5f);
		sumg -= _tmp.x;
		sumgg -= _tmp.y;
		pixNum -= tmp.z;
		*((int4*)((char*)sumG + y * pitchI4) + x) = make_int4(sumg, sumgg, pixNum, sumg / pixNum);
	}
}


__global__ void
de_calculateDC(int width_, int height_, int* G, int4* sumG, float eps2, float2* dxcx, size_t pitchI1, size_t pitchI4, size_t pitchF2)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < 0 || x >= width_ || y < 0 || y >= height_)
		return;

	int4 tmp = *((int4*)((char*)sumG + y * pitchI4) + x);
	int g = *((int*)((char*)G + y * pitchI1) + x);
	float pixel_sum_window_inv = 1.0f / (float)tmp.z;
	float g_ave = ((float)tmp.x) * pixel_sum_window_inv;
	float gg_ave = ((float)tmp.y) * pixel_sum_window_inv;
	float vx = gg_ave - g_ave * g_ave + eps2;
	float tmp2 = ((float)g) - g_ave;
	float cx2 = tmp2 * pixel_sum_window_inv / vx;
	*((float2*)((char*)dxcx + y * pitchF2) + x) = make_float2(cx2, pixel_sum_window_inv - g_ave * cx2);
}


#endif

}