
#include <hip/hip_runtime.h>

namespace FGMF_GPU_Or
{

/*
	__device__ inline int
		de_calculateHcum(const int& W_f_cum, const int* W_g_cum, const float* c_x, const float& d_x, const int channelNum_g)
	{
		float sum = d_x * W_f_cum;
		for (int i = 0; i < channelNum_g; i++)
			sum += c_x[i] * W_g_cum[i];
		return sum;
	}

	__device__ inline int
		searchWeightedMedian(const int* W_F, const int** W_G, int& W_f_cum, int** W_g_cum, int W_k, float** c_x, float& d_x, int channelNum_g)
	{
		float h = de_calculateHcum(W_f_cum, W_g_cum, c_x, d_x, channelNum_g);

		const int flagA = h < 0.5f;
		const int flag2 = flagA - 1;
		const int sign = flagA * 2 - 1;

		while (true)
		{
			W_k += flagA;
			//if(histogram[W_k].x)//ここはコメント外してもいいかも
			{
				W_f_cum += W_F[W_k] * sign;
				for (int i = 0; i < channelNum_g; i++)
					W_g_cum[i] += W_G[W_k][i] * sign;

				h = de_calculateHcum(W_f_cum, W_g_cum, c_x, d_x, channelNum_g);

				if ((h >= 0.5f) == flagA)
				{
					int result_center = W_k;
					W_k += flag2;
					return result_center;
				}
			}
			W_k += flag2;
		}
	}

	__device__ inline void
	de_addPixelToWindow(const int& f_x, const int* g_x, int* W_F, int** W_G, const int& W_f_cum, const int** W_g_cum, const int W_k, const int channelNum_g)
	{
		atomicAdd(&W_F[f_x], 1);
		for (int i = 0; i < channelNum_g; i++)
			atomicAdd(&W_G[f_x][i], g_x[i]);
		if (f_x <= W_k)
		{
			atomicAdd(&W_f_cum, 1);
			for (int i = 0; i < channelNum_g; i++)
				atomicAdd(&W_g_cum[i], g_x[i]);
		}
	}

	__device__ inline void
		de_removePixelFromWindow(const int& f_x, const int* g_x, int* W_F, int** W_G, const int channelNum_g)
	{

	}

	//gX
	__global__ void
		de_filter2D(int width, int height, int radius, int Imax, int* result_center, cudaTextureObject_t texF, cudaTextureObject_t* texGX, float** CxDx, size_t pitchI1, size_t pitchF1, int channelNum_g)
	{
		int s_center = blockIdx.x;
		if (s_center >= width || threadIdx.x >= radius * 2 + 1)
			return;
		int s = blockIdx.x + threadIdx.x - radius;


		__shared__ int W_k;
		__shared__ int W_f_cum;
		extern __shared__ int sdata[];//sizeof(int) * (1 + Imax) * channelNum_g
		int* W_F = &sdata[0];//sizeof(int) * Imax
		int* W_G = &sdata[Imax];//sizeof(int) * Imax * channelNum_g
		int* W_g_cum = &sdata[Imax * (1 + channelNum_g)];//sizeof(int) * channelNum_g

		int f;
		int* g = new int[channelNum_g];
		float* cx = new int[channelNum_g];
		float dx;

		//中心スレッドのみ実行
		if (threadIdx.x == radius)
		{
			//Initialize W
			//ヒストグラム初期化
			for (int i = 0; i < (1 + Imax) * channelNum_g; i++)
				sdata[i] = 0;

			W_k = tex2D<int>(texF, s, 0);//current index
		}
		//thread同期
		__syncthreads();

		//1つ目ヒストグラム形成
		for (int yy = -radius; yy <= radius; yy++)
		{
			//x方向のヒストグラム形成は各スレッドが担当する
			{
				f = tex2D<int>(texF, s, yy);
				for (int i = 0; i < channelNum_g; i++)
					g[i] = tex2D<int>(texGX[i], s, yy);
				for (int i = 0; i < channelNum_g; i++)
					atomicAdd(&histogramX[f * k + i], g[i]);
				atomicAdd(&histogramX[f * k + n], 1);
				if (f <= W_k)
				{
					for (int i = 0; i < channelNum_g; i++)
						atomicAdd(&fgXSumUpToIndex[i], g[i]);
					atomicAdd(&fgXSumUpToIndex[n], 1);
				}

			}
		}

		//thread同期
		__syncthreads();


		//1行目の中央値計算
		//中心スレッドのみ実行
		if (threadIdx.x == radius)
		{
			for (int i = 0; i < channelNum_g; i++)
//				cxdx[i] = *((float*)((char*)CxDx[i]) + x);
//			*((int*)((char*)result_center) + x) = de_findMedian(cxdx, histogramX, fgXSumUpToIndex, W_k, n);
		}
		//thread同期
		__syncthreads();

		//2行目以降の処理
		for (int t = 1; t < height; t++)
		{
			int tp = t + radius;
			int tm = t - radius - 1;
			//Add pixel at(s, t+) to W(2)
			f = tex2D<int>(texF, s, tp);
			for (int i = 0; i < channelNum_g; i++)
				g[i] = tex2D<int>(texGX[i], s, tp);
			de_addPixelToWindow(f, g, W_F, W_G, W_f_cum, W_g_cum, W_k, channelNum_g);

			//Remove pixel at(s, t-) from W(2)
			f = tex2D<int>(texF, s, tm);
			for (int i = 0; i < channelNum_g; i++)
				g[i] = tex2D<int>(texGX[i], s, tm);
			de_removePixelFromWindow(f, g, W_F, W_G, channelNum_g);

			//thread同期
			__syncthreads();

			
			//中心スレッドのみ実行
			if (threadIdx.x == radius)
			{
				for (int i = 0; i < k; i++)
					cxdx[i] = *((float*)((char*)CxDx[i] + t * pitchF1) + x);
				//中央値計算
				*((int*)((char*)result_center + t * pitchI1) + x) = de_findMedian(cxdx, histogramX, fgXSumUpToIndex, W_k, n);
			}
			
			//thread同期
			__syncthreads();
		}


		delete g;
		if (threadIdx.x == radius)
		{
			//delete cxdx;
		}
		__syncthreads();
	}
	*/

}